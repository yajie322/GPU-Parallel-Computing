#include "hip/hip_runtime.h"
/*
	Given a array of integers, this piece of code examines several methods that compute the
	cumulative sums of the array, i.e. in the resuting array, the n-th element is the sum of 1st
	through n-th elements in the original array.
*/

#include <stdio.h>
#include <stdlib.h>
#include "timerc.h"

#define N 1024*128
#define THREADSPERBLOCK 1024

__global__ void cumulative_sum(int *a, int *b) {
	int size = 2 * blockDim.x;
	int start = 2 * blockDim.x * blockIdx.x;

	for (int step = 1; step < size; step *= 2) {
		if (threadIdx.x < blockDim.x / step) {
			a[start + 2 * step - 1 + threadIdx.x * step * 2] +=
				a[start + step - 1 + threadIdx.x * step * 2];
		}
		__syncthreads();
	}

	for (int step = size / 2; step > 1; step /= 2) {
		if (threadIdx.x < (size / step - 1)) {
			a[start + step - 1 + step / 2 + threadIdx.x * step] +=
				a[start + step - 1 + threadIdx.x * step];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		b[blockIdx.x] = a[start + size - 1];
	}
}

__global__ void sum_blocks(int *a) {
	int size = 2 * blockDim.x;
	int start = 2 * blockDim.x * blockIdx.x;

	for (int step = 1; step < size; step *= 2) {
		if (threadIdx.x < blockDim.x / step) {
			a[start + 2 * step - 1 + threadIdx.x * step * 2] +=
				a[start + step - 1 + threadIdx.x * step * 2];
		}
		__syncthreads();
	}

	for (int step = size / 2; step > 1; step /= 2) {
		if (threadIdx.x < (size / step - 1)) {
			a[start + step - 1 + step / 2 + threadIdx.x * step] +=
				a[start + step - 1 + threadIdx.x * step];
		}
		__syncthreads();
	}
}

__global__ void fix_sum(int *a, int *b, int size) {
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if (id >= size) {
		a[id] += b[blockIdx.x/2 - 1];
	}
}

__global__ void cumulative_sum_shared_mem(int *a, int *b) {
	int size = 2 * blockDim.x;
	__shared__ int tmp[2 * THREADSPERBLOCK];

	int id = threadIdx.x + blockDim.x * blockIdx.x;
	tmp[2 * threadIdx.x] = a[2 * id];
	tmp[2 * threadIdx.x + 1] = a[2 * id + 1];
	__syncthreads();

	for (int step = 1; step < size; step *= 2) {
		if (threadIdx.x < blockDim.x / step) {
			tmp[2 * step - 1 + threadIdx.x * step * 2] +=
				tmp[step - 1 + threadIdx.x * step * 2];
		}
		__syncthreads();
	}

	for (int step = size / 2; step > 1; step /= 2) {
		if (threadIdx.x < (size / step - 1)) {
			tmp[step - 1 + step / 2 + threadIdx.x * step] +=
				tmp[step - 1 + threadIdx.x * step];
		}
		__syncthreads();
	}

	a[2 * id] = tmp[2 * threadIdx.x];
	a[2 * id + 1] = tmp[2 * threadIdx.x + 1];
	__syncthreads();

	if (threadIdx.x == 0) {
		b[blockIdx.x] = tmp[size - 1];
	}
}

__global__ void cumulative_sum_2(int *a, int *b) {
	int start = 2 * blockDim.x * blockIdx.x;
	int size = 2 * blockDim.x;

	for (int step = 1; step <= blockDim.x; step *= 2) {
		a[start + step + 2 * step * (threadIdx.x / step) + threadIdx.x % step]
			+= a[start + step + 2 * step * (threadIdx.x / step) - 1];
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		b[blockIdx.x] = a[start + size - 1];
	}
}


int main() {
	float time;
	int *dev_arr, *dev_output;
	int *host_arr = (int *) malloc(N * sizeof(int));
	int *host_output = (int *) malloc(N * sizeof(int));
	
	for (int i = 0; i < N; i++) {
		host_arr[i] = 1;
	}

	cstart();
	host_output[0] = host_arr[0];
	for (int i = 1; i < N; i++) {
		host_output[i] = host_output[i-1] + host_arr[i];
	}
	cend(&time);
	printf("cpu time = %f\n", time);

	hipMalloc((void **) &dev_arr, N * sizeof(int));
	hipMalloc((void **) &dev_output, N * sizeof(int));
	hipMemcpy(dev_arr, host_arr, N * sizeof(int), hipMemcpyHostToDevice);

	gstart();
	cumulative_sum<<<N/THREADSPERBLOCK/2,THREADSPERBLOCK>>>(dev_arr, dev_output);
	sum_blocks<<<1, 1024>>>(dev_output);
	fix_sum<<<N/THREADSPERBLOCK, THREADSPERBLOCK>>>(dev_arr, dev_output, 2 * THREADSPERBLOCK);
	gend(&time);
	printf("gpu time = %f\n", time);

	hipMemcpy(host_output, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);
	// for (int i = 0; i < N; i++) {
	// 	printf("%d ", host_output[i]);
	// }
	// printf("\n");

	hipMemcpy(dev_arr, host_arr, N * sizeof(int), hipMemcpyHostToDevice);
	gstart();
	cumulative_sum_shared_mem<<<N/THREADSPERBLOCK/2, THREADSPERBLOCK>>>(dev_arr, dev_output);
	sum_blocks<<<1, 1024>>>(dev_output);
	fix_sum<<<N/THREADSPERBLOCK, THREADSPERBLOCK>>>(dev_arr, dev_output, 2 * THREADSPERBLOCK);
	gend(&time);
	printf("gpu time (with shared memory)%f\n", time);

	hipMemcpy(host_output, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);
	// for (int i = 0; i < N; i++) {
	// 	printf("%d ", host_output[i]);
	// }
	// printf("\n");

	hipMemcpy(dev_arr, host_arr, N * sizeof(int), hipMemcpyHostToDevice);
	gstart();
	cumulative_sum_2<<<N/THREADSPERBLOCK/2, THREADSPERBLOCK>>>(dev_arr, dev_output);
	sum_blocks<<<1, 1024>>>(dev_output);
	fix_sum<<<N/THREADSPERBLOCK, THREADSPERBLOCK>>>(dev_arr, dev_output, 2 * THREADSPERBLOCK);
	gend(&time);
	printf("gpu time (with 2nd scheme)%f\n", time);

	hipMemcpy(host_output, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);
	// for (int i = 0; i < N; i++) {
	// 	printf("%d ", host_output[i]);
	// }
	// printf("\n");


	return 0;
}